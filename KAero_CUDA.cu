#include "hip/hip_runtime.h"
/*******************************************************************
*   KAero_CUDA.cpp
*   KPS
*
*	Author: Kareem Omar
*	kareem.omar@uah.edu
*	https://github.com/komrad36
*
*	Last updated Feb 12, 2016
*   This application is entirely my own work.
*******************************************************************/
//
// KAero uses CUDA or CPU and operates on a satellite or body defined as a series of polygons
// in 3-D space. When supplied with air density and velocity (in the Body frame),
// it approximates the drag force and torque on the body by simulating
// collisions and accumulating impulses and angular impulses per unit time.
//
// Note that net force and torque are returned in the Body frame.
//
// This module is intended for use by a 6 DoF orbital/attitude propagator which
// calls the aer() function from its integrators to obtain forces and/or torques,
// such as KPS.
//

#include "KAero.h"

#define PAD						(0.000001)

// placeholder x-value to signal no collision occurred
#define NONE					(-9999999.0)

// threads in one dimension
//
//
#define THREADS					(16)

// wrap around a CUDA function call to check and report any errors
bool checkForCUDAError(hipError_t cuda_status) {
	if (cuda_status != hipSuccess) {
		std::cerr << "ERROR: CUDA reports " << hipGetErrorName(cuda_status) << ". Aborting." << std::endl
			<< "More info: " << hipGetErrorString(cuda_status) << std::endl;
		return true;
	}
	return false;
}

// calculate the number of compute cores in a CUDA device
// based on its major and minor version numbers
//
// used in finding the fastest available GPU on the system
int getNumCores(const int major, const int minor) {
	const std::vector<std::pair<int, int>> mapping {
		{ 20, 32 }, // Fermi GF100
		{ 21, 48 }, // Fermi GF10x
		{ 30, 192 }, // Kepler GK10x
		{ 32, 192 }, // Kepler GK10x
		{ 35, 192 }, // Kepler GK11x
		{ 37, 192 }, // Kepler GK21x
	};

	const int combined_version = major * 10 + minor;

	// all the Maxwells have 128 so far, so making that the default
	int num_cores = 128;

	for (auto&& pair : mapping) {
		if (pair.first == combined_version) {
			num_cores = pair.second;
			break;
		}
	}

	return num_cores;
}

// approximate the GFLOPS of compute-enabled CUDA devices
// and return the fastest one
bool getHighestGFLOPSDevice(int& ID) {
	hipDeviceProp_t deviceProp;
	int num_devices = 0;
	if (checkForCUDAError(hipGetDeviceCount(&num_devices))) return false;

	if (num_devices == 0) {
		std::cerr << "ERROR: CUDA reports no capable devices on this system. Aborting." << std::endl;
		return false;
	}

	int devices_prohibited = 0;
	int best_SM_arch = 0;
	for (int i = 0; i < num_devices; ++i) {
		if (checkForCUDAError(hipGetDeviceProperties(&deviceProp, i))) return false;
		if (deviceProp.computeMode == hipComputeModeProhibited) {
			++devices_prohibited;
		}
		else {
			if (deviceProp.major > 0 && deviceProp.major < 9999) {
				best_SM_arch = std::max(best_SM_arch, deviceProp.major);
			}
		}
	}

	if (devices_prohibited == num_devices) {
		std::cerr << "ERROR: CUDA reports that all CUDA devices have compute mode prohibited. Aborting." << std::endl;
		return false;
	}

	size_t best_perf = 0;
	int best_i = 0;
	for (int i = 0; i < num_devices; ++i) {
		hipGetDeviceProperties(&deviceProp, i);
		if (deviceProp.computeMode != hipComputeModeProhibited) {
			size_t num_SMs = (deviceProp.major == 9999 && deviceProp.minor == 9999) ? 1 : getNumCores(deviceProp.major, deviceProp.minor);
			size_t perf = static_cast<size_t>(deviceProp.multiProcessorCount) * num_SMs * deviceProp.clockRate;
			if (perf  > best_perf) {
				if (best_SM_arch > 2) {
					if (deviceProp.major == best_SM_arch) {
						best_perf = perf;
						best_i = i;
					}
				}
				else {
					best_perf = perf;
					best_i = i;
				}
			}
		}
	}

	std::cout << "Autoselecting CUDA device with highest GFLOPS: Device " << best_i << std::endl;

	ID = best_i;
	return true;
}

bool KAero_CUDA::init(const int cuda_device_ID) {
	if (cuda_device_ID == AUTO_SELECT) {
		if (!getHighestGFLOPSDevice(cuda_device)) return false;
	}
	else {
		cuda_device = cuda_device_ID;
	}

	if (checkForCUDAError(hipSetDevice(cuda_device))) return false;

	if (checkForCUDAError(hipMalloc(&d_N, num_poly*sizeof(vec3)))) return false;
	if (checkForCUDAError(hipMalloc(&d_precomp, num_poly*sizeof(double)))) return false;
	if (checkForCUDAError(hipMalloc(&d_min_y, num_poly*sizeof(double)))) return false;
	if (checkForCUDAError(hipMalloc(&d_max_y, num_poly*sizeof(double)))) return false;
	if (checkForCUDAError(hipMalloc(&d_min_z, num_poly*sizeof(double)))) return false;
	if (checkForCUDAError(hipMalloc(&d_max_z, num_poly*sizeof(double)))) return false;
	if (checkForCUDAError(hipMalloc(&d_totals, 4 * sizeof(double)))) return false;
	if (checkForCUDAError(hipMalloc(&d_P_rot, total_pts * sizeof(vec3)))) return false;
	if (checkForCUDAError(hipMalloc(&d_CM_R, sizeof(vec3)))) return false;

	hipDeviceProp_t device_prop;
	if (checkForCUDAError(hipGetDeviceProperties(&device_prop, cuda_device))) return false;

	std::cout << "CUDA KAero initialized on device " << cuda_device << ": " << device_prop.name << std::endl;

	return true;
}

KAero_CUDA::KAero_CUDA(const double linear_pitch, const int num_polygons, const vec3* const poly, const vec3 sat_CM) :
pitch(linear_pitch),
f_scalar(2.0*linear_pitch*linear_pitch),
num_poly(num_polygons),
total_pts(num_polygons * NUM_VTX),
CM(sat_CM),
P_s(new vec3[num_polygons * NUM_VTX]),
P_rot(new vec3[num_polygons * NUM_VTX]) {

	// copy polygon data into internal storage
	for (int i = 0; i < total_pts; ++i) {
		P_s[i] = poly[i];
	}

}

KAero_CUDA::~KAero_CUDA() {
	hipDeviceReset();
}

// kernel to precompute normals and portion of collision location finding
template<int block_size>
__global__ void precompute(vec3* __restrict__ const d_P_rot,
	vec3* __restrict__ const d_N, double* __restrict__ const d_precomp,
	double* __restrict__ const d_min_y, double* __restrict__ const d_max_y,
	double* __restrict__ const d_min_z, double* __restrict__ const d_max_z,
	double* __restrict__ const d_totals, int num_poly) {

	int i = threadIdx.x;

	// partition shared memory into 4 chunks of num_poly doublesm
	// for storing the min y's, max y's, min z's, and max z's, respectively
	extern __shared__ double s_min_y[];
	double* s_max_y = s_min_y + num_poly;
	double* s_min_z = s_min_y + 2 * num_poly;
	double* s_max_z = s_min_y + 3 * num_poly;

	double l_min_y = DBL_MAX;
	double l_max_y = -DBL_MAX;
	double l_min_z = DBL_MAX;
	double l_max_z = -DBL_MAX;

	if (i < num_poly) {
		// precompute some info for speed,
		// including panel normals and some of collision location math
		vec3* P = d_P_rot + i*NUM_VTX;
		d_N[i] = glm::normalize(glm::cross(P[2] - P[1], P[2] - P[3]));
		d_precomp[i] = glm::dot(d_N[i], P[0]);

		l_min_y = l_max_y = P[0].y;
		l_min_z = l_max_z = P[0].z;

		// find mins and maxes of y and z of each poly
#pragma unroll
		for (int j = 1; j < NUM_VTX; ++j) {
			l_min_y = (P[j].y < l_min_y) ? P[j].y : l_min_y;
			l_max_y = (P[j].y > l_max_y) ? P[j].y : l_max_y;
			l_min_z = (P[j].z < l_min_z) ? P[j].z : l_min_z;
			l_max_z = (P[j].z > l_max_z) ? P[j].z : l_max_z;
		}

		d_min_y[i] = l_min_y;
		d_max_y[i] = l_max_y;
		d_min_z[i] = l_min_z;
		d_max_z[i] = l_max_z;

	}

	s_min_y[i] = l_min_y;
	s_max_y[i] = l_max_y;
	s_min_z[i] = l_min_z;
	s_max_z[i] = l_max_z;
	__syncthreads();

	// fully unrolled reduction above 64
	// see CUDA reduction documentation or KPS reseach paper for more info

	if (block_size >= 512 && i < 256) {
		s_min_y[i] = l_min_y = s_min_y[i + 256] < l_min_y ? s_min_y[i + 256] : l_min_y;
		s_max_y[i] = l_max_y = s_max_y[i + 256] > l_max_y ? s_max_y[i + 256] : l_max_y;
		s_min_z[i] = l_min_z = s_min_z[i + 256] < l_min_z ? s_min_z[i + 256] : l_min_z;
		s_max_z[i] = l_max_z = s_max_z[i + 256] > l_max_z ? s_max_z[i + 256] : l_max_z;
	}
	__syncthreads();

	if (block_size >= 256 && i < 128) {
		s_min_y[i] = l_min_y = s_min_y[i + 128] < l_min_y ? s_min_y[i + 128] : l_min_y;
		s_max_y[i] = l_max_y = s_max_y[i + 128] > l_max_y ? s_max_y[i + 128] : l_max_y;
		s_min_z[i] = l_min_z = s_min_z[i + 128] < l_min_z ? s_min_z[i + 128] : l_min_z;
		s_max_z[i] = l_max_z = s_max_z[i + 128] > l_max_z ? s_max_z[i + 128] : l_max_z;
	}
	__syncthreads();

	if (block_size >= 128 && i < 64) {
		s_min_y[i] = l_min_y = s_min_y[i + 64] < l_min_y ? s_min_y[i + 64] : l_min_y;
		s_max_y[i] = l_max_y = s_max_y[i + 64] > l_max_y ? s_max_y[i + 64] : l_max_y;
		s_min_z[i] = l_min_z = s_min_z[i + 64] < l_min_z ? s_min_z[i + 64] : l_min_z;
		s_max_z[i] = l_max_z = s_max_z[i + 64] > l_max_z ? s_max_z[i + 64] : l_max_z;
	}
	__syncthreads();

	// at this point 64 remain
	// CUDA architectures above 3.0 support the warp shuffle operation to do the last warp
	// very efficiently.
	//
	// see CUDA warp shuffle documentation for more info
	//
	// if available, warp shuffle is used.
	// if not, the remaining steps are manually unrolled just like above

#if (__CUDA_ARCH__ >= 300)
	if (i < 32) {
		if (block_size >= 64) {
			l_min_y = s_min_y[i + 32] < l_min_y ? s_min_y[i + 32] : l_min_y;
			l_max_y = s_max_y[i + 32] > l_max_y ? s_max_y[i + 32] : l_max_y;
			l_min_z = s_min_z[i + 32] < l_min_z ? s_min_z[i + 32] : l_min_z;
			l_max_z = s_max_z[i + 32] > l_max_z ? s_max_z[i + 32] : l_max_z;
		}

		for (int offset = warpSize / 2; offset > 0; offset /= 2) {
			double shuffled_min_y = __shfl_down(l_min_y, offset);
			l_min_y = shuffled_min_y < l_min_y ? shuffled_min_y : l_min_y;

			double shuffled_max_y = __shfl_down(l_max_y, offset);
			l_max_y = shuffled_max_y > l_max_y ? shuffled_max_y : l_max_y;

			double shuffled_min_z = __shfl_down(l_min_z, offset);
			l_min_z = shuffled_min_z < l_min_z ? shuffled_min_z : l_min_z;

			double shuffled_max_z = __shfl_down(l_max_z, offset);
			l_max_z = shuffled_max_z > l_max_z ? shuffled_max_z : l_max_z;
		}
	}

#else
	// fully unrolled reduction within single warp
	if (block_size >= 64 && i < 32)
	{
		s_min_y[i] = l_min_y = s_min_y[i + 32] < l_min_y ? s_min_y[i + 32] : l_min_y;
		s_max_y[i] = l_max_y = s_max_y[i + 32] > l_max_y ? s_max_y[i + 32] : l_max_y;
		s_min_z[i] = l_min_z = s_min_z[i + 32] < l_min_z ? s_min_z[i + 32] : l_min_z;
		s_max_z[i] = l_max_z = s_max_z[i + 32] > l_max_z ? s_max_z[i + 32] : l_max_z;
	}
	__syncthreads();

	if (block_size >= 32 && i < 16)
	{
		s_min_y[i] = l_min_y = s_min_y[i + 16] < l_min_y ? s_min_y[i + 16] : l_min_y;
		s_max_y[i] = l_max_y = s_max_y[i + 16] > l_max_y ? s_max_y[i + 16] : l_max_y;
		s_min_z[i] = l_min_z = s_min_z[i + 16] < l_min_z ? s_min_z[i + 16] : l_min_z;
		s_max_z[i] = l_max_z = s_max_z[i + 16] > l_max_z ? s_max_z[i + 16] : l_max_z;
	}
	__syncthreads();

	if (block_size >= 16 && i < 8)
	{
		s_min_y[i] = l_min_y = s_min_y[i + 8] < l_min_y ? s_min_y[i + 8] : l_min_y;
		s_max_y[i] = l_max_y = s_max_y[i + 8] > l_max_y ? s_max_y[i + 8] : l_max_y;
		s_min_z[i] = l_min_z = s_min_z[i + 8] < l_min_z ? s_min_z[i + 8] : l_min_z;
		s_max_z[i] = l_max_z = s_max_z[i + 8] > l_max_z ? s_max_z[i + 8] : l_max_z;
	}
	__syncthreads();

	if (block_size >= 8 && i < 4)
	{
		s_min_y[i] = l_min_y = s_min_y[i + 4] < l_min_y ? s_min_y[i + 4] : l_min_y;
		s_max_y[i] = l_max_y = s_max_y[i + 4] > l_max_y ? s_max_y[i + 4] : l_max_y;
		s_min_z[i] = l_min_z = s_min_z[i + 4] < l_min_z ? s_min_z[i + 4] : l_min_z;
		s_max_z[i] = l_max_z = s_max_z[i + 4] > l_max_z ? s_max_z[i + 4] : l_max_z;
	}
	__syncthreads();

	if (block_size >= 4 && i < 2)
	{
		s_min_y[i] = l_min_y = s_min_y[i + 2] < l_min_y ? s_min_y[i + 2] : l_min_y;
		s_max_y[i] = l_max_y = s_max_y[i + 2] > l_max_y ? s_max_y[i + 2] : l_max_y;
		s_min_z[i] = l_min_z = s_min_z[i + 2] < l_min_z ? s_min_z[i + 2] : l_min_z;
		s_max_z[i] = l_max_z = s_max_z[i + 2] > l_max_z ? s_max_z[i + 2] : l_max_z;
	}
	__syncthreads();

	if (block_size >= 2 && i < 1)
	{
		s_min_y[i] = l_min_y = s_min_y[i + 1] < l_min_y ? s_min_y[i + 1] : l_min_y;
		s_max_y[i] = l_max_y = s_max_y[i + 1] > l_max_y ? s_max_y[i + 1] : l_max_y;
		s_min_z[i] = l_min_z = s_min_z[i + 1] < l_min_z ? s_min_z[i + 1] : l_min_z;
		s_max_z[i] = l_max_z = s_max_z[i + 1] > l_max_z ? s_max_z[i + 1] : l_max_z;
	}
	__syncthreads();
#endif

	// thread 0 contains this block's final result
	// store it out to global mem
	if (i == 0) {
		d_totals[0] = l_min_y;
		d_totals[1] = l_max_y;
		d_totals[2] = l_min_z;
		d_totals[3] = l_max_z;
	}

}

// kernel to collide particles and determine resultant forces and/or torques
__global__ void collide(vec3* __restrict__ const d_P_rot,
	vec3* __restrict__ const d_N, double* __restrict__ const d_precomp,
	double* __restrict__ const d_min_y, double* __restrict__ const d_max_y,
	double* __restrict__ const d_min_z, double* __restrict__ const d_max_z,
	int num_k, int num_m, int min_k, int min_m, double rho, double v_mag2,
	vec3* __restrict__ const d_CM_R, vec3* __restrict__ const d_block_sums_f,
	vec3* __restrict__ const d_block_sums_t, double pitch,
	int num_poly, double f_scalar) {

	int tidx = THREADS*blockIdx.x + threadIdx.x;
	int tidy = THREADS*blockIdx.y + threadIdx.y;

	// partition shared memory into forces and torques
	extern __shared__ vec3 sdata_f[];
	vec3* sdata_t = sdata_f + THREADS*THREADS;

	int sid = threadIdx.y*THREADS + threadIdx.x;

	vec3 force, torque;

	if (tidx < num_k && tidy < num_m) {
		double y = (min_k + tidx) * pitch;
		double z = (min_m + tidy) * pitch;

		double best_x = NONE;
		vec3 best_N;

		// check for collisions against each poly:
		for (int p = 0; p < num_poly; ++p) {
			// bail early if outside bounding box of this particular poly...
			if (y < d_min_y[p] || y > d_max_y[p] || z < d_min_z[p] || z > d_max_z[p]) continue;

			// ...otherwise, perform point-in-polygon anaylsis.
			// Looks scary but isn't too bad. Pretend coordinate system
			// is such that test point is at origin. Send the point to the right (+x)
			// and flip a flag each time it crosses a segment of the polygon. Flag will
			// end up flipped if inside polygon because it will cross an odd number of
			// segments. This works even for concave polygons.
			// For each segment of the polygon:
			//	if both ends are above or below x axis (i.e. share same sign), not a crossing
			//		otherwise, if both ends are left of the y axis, not a crossing
			//			otherwise, we do the slow bit (but usually don't have to due to the above):
			//			see if the segment intersects the x axis right of 0. if it does, crossing!
			// See KPS research paper for more.
			vec3* P = d_P_rot + p*NUM_VTX;
			int j = NUM_VTX - 1;
			int odd_nodes = 0;

#pragma unroll
			for (int i = 0; i < NUM_VTX; ++i) {
				odd_nodes ^= (((((P[i].z < z && P[j].z >= z) || (P[j].z < z && P[i].z >= z))
					&& (P[i].y <= y || P[j].y <= y)))
					&& ((P[i].y + (z - P[i].z) / (P[j].z - P[i].z)*(P[j].y - P[i].y) < y)));
				j = i;
			}

			// if inside polygon, compute collision location
			if (odd_nodes && d_N[p].x) {
				double x = (d_precomp[p] - (d_N[p].y*y + d_N[p].z*z)) / d_N[p].x;

				// and if it's the best (first) collision, update best_x
				if (x > best_x) {
					best_x = x;
					best_N = d_N[p];
				}
			}
		}

		// if a collision occurred
		if (best_x > NONE + PAD) {
			// see Equation 62 in KPS Research paper
			force = f_scalar*rho*best_N*(-v_mag2*best_N.x*fabs(best_N.x));

			// see Equation 63 in KPS Research paper
			torque = glm::cross(vec3{ best_x, y, z } -(*d_CM_R), force);
		}
	}

	sdata_f[sid] = force;
	sdata_t[sid] = torque;
	__syncthreads();

	// sum reduction in shared mem (block size is known to be 256)
	if (sid < 128) {
		sdata_f[sid] = force = force + sdata_f[sid + 128];
		sdata_t[sid] = torque = torque + sdata_t[sid + 128];
	}
	__syncthreads();

	if (sid < 64) {
		sdata_f[sid] = force = force + sdata_f[sid + 64];
		sdata_t[sid] = torque = torque + sdata_t[sid + 64];
	}
	__syncthreads();

	// at this point 64 remain
	// CUDA architectures above 3.0 support the warp shuffle operation to do the last warp
	// very efficiently.
	//
	// see CUDA warp shuffle documentation for more info
	//
	// if available, warp shuffle is used.
	// if not, the remaining steps are manually unrolled just like above

#if (__CUDA_ARCH__ >= 300)
	if (sid < 32) {
		force += sdata_f[sid + 32];
		torque += sdata_t[sid + 32];

		for (int offset = warpSize / 2; offset > 0; offset /= 2) {
			force.x += __shfl_down(force.x, offset);
			force.y += __shfl_down(force.y, offset);
			force.z += __shfl_down(force.z, offset);

			torque.x += __shfl_down(torque.x, offset);
			torque.y += __shfl_down(torque.y, offset);
			torque.z += __shfl_down(torque.z, offset);
		}

	}
#else
	// fully unrolled reduction within single warp
	if (sid < 32) {
		sdata_f[sid] = force = force + sdata_f[sid + 32];
		sdata_t[sid] = torque = torque + sdata_t[sid + 32];
	}
	__syncthreads();

	if (sid < 16) {
		sdata_f[sid] = force = force + sdata_f[sid + 16];
		sdata_t[sid] = torque = torque + sdata_t[sid + 16];
	}
	__syncthreads();

	if (sid < 8) {
		sdata_f[sid] = force = force + sdata_f[sid + 8];
		sdata_t[sid] = torque = torque + sdata_t[sid + 8];
	}
	__syncthreads();

	if (sid < 4) {
		sdata_f[sid] = force = force + sdata_f[sid + 4];
		sdata_t[sid] = torque = torque + sdata_t[sid + 4];
	}
	__syncthreads();

	if (sid < 2) {
		sdata_f[sid] = force = force + sdata_f[sid + 2];
		sdata_t[sid] = torque = torque + sdata_t[sid + 2];
	}

	__syncthreads();

	if (sid < 1) {
		sdata_f[sid] = force = force + sdata_f[sid + 1];
		sdata_t[sid] = torque = torque + sdata_t[sid + 1];
	}
	__syncthreads();
#endif

	// thread 0 contains this block's final result
	// store it out to global mem
	if (sid == 0) {
		d_block_sums_f[blockIdx.x*gridDim.y + blockIdx.y] = force;
		d_block_sums_t[blockIdx.x*gridDim.y + blockIdx.y] = torque;
	}

}

void KAero_CUDA::aer(vec3& f, vec3& t, const double rho, const vec3& v) {

	// zero out 'f' and 't'
	f = t = vec3();

	double v_mag2 = glm::length2(v);
	double v_mag = sqrt(v_mag2);

	// --- ROTATOR SETUP ---
	// rotates entire satellite such that velocity is in +x in new frame, i.e. relative wind
	// direction is -x
	//
	// Uses Rodrigues' rotation formula for speed
	// the ternary handles cases where velocity is entirely in +x or entirely in -x already
	//
	// does NOT handle zero velocity, but if that's the case in an *orbital simulation*
	// you have bigger problems to worry about
	double cos_theta = v.x / v_mag;
	double sin_theta = sin(acos(cos_theta));
	vec3 k = sin_theta ? vec3{ 0.0, v.z, -v.y } / (v_mag * sin_theta) : vec3();
	vec3 k_times_1_minus_cos_theta = k*(1 - cos_theta);
	// --- /ROTATOR SETUP ---

	// Use of ROTATOR (Rodrigues' formula) to rotate satellite CM into new frame
	vec3 CM_R = cos_theta*CM + sin_theta*glm::cross(k, CM) + k_times_1_minus_cos_theta*glm::dot(k, CM);

	// push CM to GPU
	// NOTE that no error checks are being done on CUDA returns;
	// they are during initialization, but not in the hot path
	hipMemcpy(d_CM_R, &CM_R, sizeof(vec3), hipMemcpyHostToDevice);

	for (int i = 0; i < total_pts; ++i) {
		// Use of ROTATOR (Rodrigues' formula) to rotate polygons into new frame
		P_rot[i] = cos_theta*P_s[i] + sin_theta*glm::cross(k, P_s[i]) + k_times_1_minus_cos_theta*glm::dot(k, P_s[i]);
	}

	// push rotated polygons to GPU (proved faster than doing the rotation on GPU)
	hipMemcpy(d_P_rot, P_rot, total_pts*sizeof(vec3), hipMemcpyHostToDevice);


	// --- PRECOMPUTE ---
	// precomp includes a reduction for total satellite bounding box,
	// so launch a power of 2 number of threads
	//
	// see KPS reseach paper and/or CUDA documentation on reductions for more info
	int precomp_threads = nextHigherPow2(num_poly);

	// increase kernel performance by templating on thread count
	// polygon number doesn't change that much... currently supporting < 256
	//
	// Blocks: 1
	// Threads: next power of two above num_poly
	// Shared memory: 4 doubles (min_y, max_y, min_z, max_z) for each thread,
	//				EXCEPT not less than 64 total groups of 4 doubles
	//				because 2 warps (of 32) are combined before shuffling begins
	//				within each warp
	//
	// These switch statements are out of order intentionally; they are arranged
	// in order of most likelihood to assist the branch predictor
	// (usually ~10 polygons, so 16 threads... sometimes ~6 polygons, so 8, etc...)
	//
	switch (precomp_threads) {
	case 16:
		precompute<16> << <1, 16, 4 * 64 * sizeof(double) >> >(d_P_rot, d_N,
			d_precomp, d_min_y, d_max_y, d_min_z, d_max_z, d_totals, num_poly);
		break;
	case 8:
		precompute<8> << <1, 8, 4 * 64 * sizeof(double) >> >(d_P_rot, d_N,
			d_precomp, d_min_y, d_max_y, d_min_z, d_max_z, d_totals, num_poly);
		break;
	case 4:
		precompute<4> << <1, 4, 4 * 64 * sizeof(double) >> >(d_P_rot, d_N,
			d_precomp, d_min_y, d_max_y, d_min_z, d_max_z, d_totals, num_poly);
		break;
	case 32:
		precompute<32> << <1, 32, 4 * 64 * sizeof(double) >> >(d_P_rot, d_N,
			d_precomp, d_min_y, d_max_y, d_min_z, d_max_z, d_totals, num_poly);
		break;
	case 2:
		precompute<2> << <1, 2, 4 * 64 * sizeof(double) >> >(d_P_rot, d_N,
			d_precomp, d_min_y, d_max_y, d_min_z, d_max_z, d_totals, num_poly);
		break;
	case 64:
		precompute<64> << <1, 64, 4 * 64 * sizeof(double) >> >(d_P_rot, d_N,
			d_precomp, d_min_y, d_max_y, d_min_z, d_max_z, d_totals, num_poly);
		break;
	case 1:
		precompute<1> << <1, 1, 4 * 64 * sizeof(double) >> >(d_P_rot, d_N,
			d_precomp, d_min_y, d_max_y, d_min_z, d_max_z, d_totals, num_poly);
		break;
	case 128:
		precompute<128> << <1, 128, 4 * 128 * sizeof(double) >> >(d_P_rot, d_N,
			d_precomp, d_min_y, d_max_y, d_min_z, d_max_z, d_totals, num_poly);
		break;
	case 256:
		precompute<256> << <1, 256, 4 * 256 * sizeof(double) >> >(d_P_rot, d_N,
			d_precomp, d_min_y, d_max_y, d_min_z, d_max_z, d_totals, num_poly);
		break;
	case 512:
		precompute<512> << <1, 512, 4 * 512 * sizeof(double) >> >(d_P_rot, d_N,
			d_precomp, d_min_y, d_max_y, d_min_z, d_max_z, d_totals, num_poly);
		break;
	default:
		std::cerr
			<< "ERROR: Too many polygons! This should never occur, as it should be caught" << std::endl
			<< "in the polygon input stage." << std::endl;
		break;
	}

	hipMemcpy(totals, d_totals, 4 * sizeof(double), hipMemcpyDeviceToHost);
	// --- /PRECOMPUTE ---


	// CPU: use total bounding box information to get rectangular area to simulate
	const int min_k = static_cast<int>((totals[0] + PAD) / pitch);
	const int max_k = static_cast<int>((totals[1] - PAD) / pitch);
	const int min_m = static_cast<int>((totals[2] + PAD) / pitch);
	const int max_m = static_cast<int>((totals[3] - PAD) / pitch);

	// determine number of particles required in each direction
	const int num_k = max_k - min_k + 1;
	const int num_m = max_m - min_m + 1;

	// determine required number of collide blocks in 2-D grid
	const int collide_blks_x = (num_k - 1) / THREADS + 1;
	const int collide_blks_y = (num_m - 1) / THREADS + 1;

	const int collide_blks_total = collide_blks_x * collide_blks_y;

	// dynamically allocate device memory for the force and torque sums from each block
	hipMalloc(&d_block_sums_f, collide_blks_total*sizeof(vec3));
	hipMalloc(&d_block_sums_t, collide_blks_total*sizeof(vec3));

	// dynamically allocate host memory for the force and torque sums from each block
	block_sums_f = new vec3[collide_blks_total];
	block_sums_t = new vec3[collide_blks_total];

	// collide!
	// shared memory is 2 vec3's (one for force, one for torque) per thread
	// NOTE that collide also performs a reduction, but the thread count is fixed at THREADS, which is 16,
	// already a multiple of 2.
	collide<<<dim3(collide_blks_x, collide_blks_y), dim3(THREADS, THREADS), 2*THREADS*THREADS*sizeof(vec3)>>>(d_P_rot,
		d_N, d_precomp, d_min_y, d_max_y, d_min_z, d_max_z, num_k, num_m, min_k, min_m, rho, v_mag2,
		d_CM_R, d_block_sums_f, d_block_sums_t, pitch, num_poly, f_scalar);

	// retrieve force block sums
	hipMemcpy(block_sums_f, d_block_sums_f, collide_blks_total*sizeof(vec3), hipMemcpyDeviceToHost);

	// free force block sum device dynamic allocation
	hipFree(d_block_sums_f);

	// retrieve torque block sums
	hipMemcpy(block_sums_t, d_block_sums_t, collide_blks_total*sizeof(vec3), hipMemcpyDeviceToHost);

	// free torque block sum device dynamic allocation
	hipFree(d_block_sums_t);

	// accumulate the block sums
	// there won't be too many; Kahan summation is not required
	for (int i = 0; i < collide_blks_total; ++i) {
		f += block_sums_f[i];
		t += block_sums_t[i];
	}

	// Use of ROTATOR (Rodrigues' formula) to rotate summed force and torque BACK TO BODY FRAME
	// (note the negative sign)
	f = cos_theta*f - sin_theta*glm::cross(k, f) + k_times_1_minus_cos_theta*glm::dot(k, f);
	t = cos_theta*t - sin_theta*glm::cross(k, t) + k_times_1_minus_cos_theta*glm::dot(k, t);

	// free force block sum host dynamic allocation
	delete[] block_sums_f;

	// free torque block sum host dynamic allocation
	delete[] block_sums_t;

}
